#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

#define CUDA_CHECK_ERRORS()                                           \
  do {                                                                \
    hipError_t err = hipGetLastError();                             \
    if (hipSuccess != err) {                                         \
      fprintf(stderr, "CUDA kernel failed : %s\n%s at L:%d in %s\n",  \
              hipGetErrorString(err), __PRETTY_FUNCTION__, __LINE__, \
              __FILE__);                                              \
      exit(-1);                                                       \
    }                                                                 \
  } while (0)

#define TOTAL_THREADS 512
inline int opt_n_threads(int work_size) {
  const int pow_2 = std::log(static_cast<double>(work_size)) / std::log(2.0);

  return max(min(1 << pow_2, TOTAL_THREADS), 1);
}


// input: query(b, m, 3) source(b, n, 3)
// output: idx(b, m, nsample)
__global__ void ball_query_kernel(int b, int n, int m, float radius,
                                  int nsample,
                                  const float *__restrict__ query,
                                  const float *__restrict__ source,
                                  int *__restrict__ idx) {
  int batch_index = blockIdx.x;
  source += batch_index * n * 3;
  query += batch_index * m * 3;
  idx += m * nsample * batch_index;

  int index = threadIdx.x;
  int stride = blockDim.x;

  float radius2 = radius * radius;
  for (int j = index; j < m; j += stride) {
    float query_x = query[j * 3 + 0];
    float query_y = query[j * 3 + 1];
    float query_z = query[j * 3 + 2];
    for (int k = 0, cnt = 0; k < n && cnt < nsample; ++k) {
      float source_x = source[k * 3 + 0];
      float source_y = source[k * 3 + 1];
      float source_z = source[k * 3 + 2];
      float d2 = (query_x - source_x) * (query_x - source_x) + 
			     (query_y - source_y) * (query_y - source_y) + 
				 (query_z - source_z) * (query_z - source_z);
      if (d2 < radius2) {
        //if (cnt == 0) {
        //  for (int l = 0; l < nsample; ++l) {
        //    idx[j * nsample + l] = k;
        //  }
        //}
        idx[j * nsample + cnt] = k;
        ++cnt;
      }
    }
  }
}


void ball_query_wrapper(int b, int n, int m, float radius,
                        int nsample, const float *query,
                        const float *source, int *idx) {
  ball_query_kernel<<<b, opt_n_threads(m), 0>>>(
      b, n, m, radius, nsample, query, source, idx);

  CUDA_CHECK_ERRORS();
}
